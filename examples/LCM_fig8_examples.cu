#include "hip/hip_runtime.h"
/***
nvcc -std=c++11 -o fig8.exe LCM_fig8_examples.cu ../utils/cudaUtils.cu ../utils/threadUtils.cpp -llcm -gencode arch=compute_61,code=sm_61 -O3
***/
#define USE_WAFR_URDF 0
#define EE_COST 1
#define USE_SMOOTH_ABS 0
#define USE_EE_VEL_COST 0
#define USE_LIMITS_FLAG 0

#define MPC_MODE 1
#define USE_LCM 1
#define USE_VELOCITY_FILTER 0
#define HARDWARE_MODE 1
#define USE_ALG_TRACE 0
#define USE_MAX_SOLVER_TIME 0
#define USE_FEEDBACK_IN_TRAJ_RUNNER 1
#define TRAJ_RUNNER_TIME_STEPS NUM_TIME_STEPS/4
#define PD_GAINS_ON_STATE 0

#define IGNORE_MAX_ROX_EXIT 0
#define TOL_COST 0.00001
#define SOLVES_TO_RESET 15
#define PLANT 4

#define E_NORM_LIM 0.05
#define V_NORM_LIM 0.05
#define TRAJ_RUNNER_ALPHA 0 // smoothing on torque and pos commands per command

#if USE_EE_VEL_COST
	// default cost terms for the start of the goal to drop the arm from the initial point to the start of the fig 8
	// delta xyz, delta rpy, u, xzyrpyd, xyzrpy
	#define SMALL 0//0.00001
	#define _Q_EE1 50.0
	#define _Q_EE2 SMALL
	#define _R_EE 0.001
	#define _QF_EE1 100.0
	#define _QF_EE2 SMALL
	#define _Q_xdEE 10.0
	#define _QF_xdEE 10.0
	#define _Q_xEE SMALL
	#define _QF_xEE SMALL
	#define _Q_EEV1 0.0
	#define _Q_EEV2 0.0
	#define _QF_EEV1 0.0
	#define _QF_EEV2 0.0
	// new cost terms for the actual fig 8 tracking
	#define _Q_EE1_fig8 300.0
	#define _Q_EE2_fig8 SMALL
	#define _R_EE_fig8 0.0005 // make 0.001 for the move to inital goal and then to 0.0005 for motion
	#define _QF_EE1_fig8 300.0
	#define _QF_EE2_fig8 SMALL
	#define _Q_xdEE_fig8 10.0
	#define _QF_xdEE_fig8 10.0
	#define _Q_xEE_fig8 1.0
	#define _QF_xEE_fig8 1.0
	#define _Q_EEV1_fig8 0
	#define _Q_EEV2_fig8 0
	#define _QF_EEV1_fig8 0
	#define _QF_EEV2_fig8 0
#else
	// default cost terms for the start of the goal to drop the arm from the initial point to the start of the fig 8
	// delta xyz, delta rpy, u, xzyrpyd, xyzrpy
	#define SMALL 0//0.00001
	#define _Q_EE1 50.0
	#define _Q_EE2 SMALL
	#define _R_EE 0.001
	#define _QF_EE1 100.0
	#define _QF_EE2 SMALL
	#define _Q_xdEE 10.0
	#define _QF_xdEE 10.0
	#define _Q_xEE SMALL
	#define _QF_xEE SMALL
	// new cost terms for the actual fig 8 tracking
	#define _Q_EE1_fig8 300.0
	#define _Q_EE2_fig8 SMALL
	#define _R_EE_fig8 0.001 // make 0.001 for the move to inital goal and then to 0.0005 for motion
	#define _QF_EE1_fig8 300.0
	#define _QF_EE2_fig8 SMALL
	#define _Q_xdEE_fig8 5.0
	#define _QF_xdEE_fig8 5.0
	#define _Q_xEE_fig8 5.0
	#define _QF_xEE_fig8 5.0
	#define _Q_EEV1_fig8 0
	#define _Q_EEV2_fig8 0
	#define _QF_EEV1_fig8 0
	#define _QF_EEV2_fig8 0
#endif

#include "../config.cuh"

template <typename T>
class LCM_Fig8Goal_Handler {
    public:
    	double totalTime;	double zeroTime;	int inFig8;
    	double eNormLim;	double vNormLim;	int costSent;
    	double totalError;	int numIters;		int currRep;
    	int iterLimit;		int timeLimit;
    	lcm::LCM lcm_ptr; // ptr to LCM object for publish ability
    	struct timeval start, end; int timeCount; double timeTotal;

    	LCM_Fig8Goal_Handler(double tTime, double eLim, double vLim, int iL, int tL) : 
    		totalTime(tTime), eNormLim(eLim), vNormLim(vLim), iterLimit(iL), timeLimit(tL) {
    		zeroTime = 0; inFig8 = 0; costSent = 0;	totalError = 0;	numIters = 0; currRep = 0;
    		if(!lcm_ptr.good()){printf("LCM Failed to Init in Goal Handler\n");}
    	}
    	~LCM_Fig8Goal_Handler(){}

    	// fig 8 goals
    	int loadFig8Goal(T *goal, double time){
			T xGoals[] = {0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004};
			T yGoals[] = {0.13686922001827645,0.1281183229143938,0.11926059413541247,0.11030432312999372,0.1012577993467988,0.09212931223448896,0.08292715124172548,0.0736596058171696,0.06433496540948258,0.05496151946732569,0.045547557439360176,0.03610136877424733,0.026631242920648335,0.017145469327224508,0.007652337442637103,-0.001839863284452653,-0.011322843405383448,-0.020788313471494096,-0.030227984034123273,-0.039633565644609764,-0.0489967688542923,-0.05830930421450961,-0.06756288227660043,-0.07674921359190354,-0.08586000871175764,-0.09488697818749958,-0.1038655258329727,-0.11272375591745916,-0.12145267616480321,-0.13004109492853494,-0.13848752180117063,-0.1467858524541864,-0.1549281455978588,-0.1629165235905652,-0.1707482796892309,-0.17841599130217783,-0.18591882035832843,-0.1932502389635062,-0.20040076105913676,-0.20735798772338723,-0.21411519853223893,-0.22066551450768562,-0.2269966386554578,-0.2331053523296066,-0.23897989082424964,-0.24461177603116246,-0.24999586174091798,-0.255127737887603,-0.2600037985005273,-0.264623973362925,-0.2689800086072113,-0.27306787696210033,-0.2768840041676995,-0.2804279779836409,-0.28369462501981374,-0.28667669709392635,-0.2893761036490012,-0.29178950599107006,-0.29391158123626243,-0.29574034972478985,-0.29727453396452347,-0.29851358128156347,-0.2994544451742904,-0.30009482660988346,-0.30043907902783173,-0.30048644363405164,-0.3002369544043975,-0.29968819678814346,-0.2988444313574429,-0.2977048064508765,-0.2962719042333858,-0.29454317198576974,-0.2925200231035754,-0.2902045731500091,-0.2876022809123594,-0.2847133796914938,-0.28154140071122247,-0.27809048676073667,-0.27436270583704647,-0.2703665745543449,-0.26610524813698555,-0.26158238012185503,-0.25680474076422527,-0.2517742032662073,-0.2464935808285531,-0.24096598110942027,-0.23519855715870805,-0.22919065210989617,-0.22294815964121067,-0.216477551020404,-0.20978904844956445,-0.20289009717093992,-0.19578529751522875,-0.18849136560284127,-0.1810102720333381,-0.17335332623615154,-0.16552932116242403,-0.15754446533330008,-0.1494055335341655,-0.141119776212369,-0.13269158757842503,-0.12412621123921029,-0.11543344583116043,-0.10661783052843572,-0.09768974155699829,-0.08866263788534111,-0.07955032956578706,-0.07036309089673469,-0.06111613853088287,-0.051824258760106816,-0.04250384833178472,-0.03316965050170948,-0.02383739609051122,-0.014517898480146497,-0.00521285872839692,0.004084069854356758,0.013377716369566438,0.022678964415954628,0.03199791841264746,0.04134623887290685,0.05072829056150665,0.06014998301207159,0.06960936374549687,0.07908892160637689,0.08855068744478754,0.09794356384062872,0.10720878615596977,0.11629326145552335,0.12514991959047528,0.13374545516319478,0.14207020091055017,0.15012393938707272,0.15792746608076888,0.16551126865840357,0.1729042968130615,0.18013571994835326,0.1872239892183252,0.19418299168723718,0.20100573965434845,0.2076839316858779,0.2142059797503866,0.22054968628960595,0.22669845070103078,0.2326348093954518,0.2383522458039774,0.24384287073319427,0.2491022174907651,0.2541293098371375,0.2589202699947854,0.2634683865999179,0.2677700650071298,0.27181899023759926,0.2756002672767167,0.2791048721375289,0.28231577838682104,0.28522354073704403,0.2878230537497963,0.2901076694828819,0.2920783437903407,0.29374310403406084,0.295107993386028,0.2961895263765827,0.2970056784978085,0.2975819271104611,0.29794495005548816,0.298109264082986,0.2980740741579594,0.2978323971097816,0.2973591836734637,0.296622924641549,0.2955794967043364,0.29418154658089085,0.2923728011959445,0.2900962060295761,0.2873037962354698,0.28396753800504204,0.28032650994085717,0.2763886650341475,0.27216195627614015,0.26765433665806226,0.2628737591711411,0.2578281768066035,0.25252554255567694,0.2469738094095883,0.2411809303595648,0.2351548583968336,0.2289035465126218,0.22243494769815644,0.21575701494466482,0.208877701243374,0.201804959585511,0.19454674296230312,0.1871110043649775,0.179505696784761,0.1717387732128811,0.16381818664056474,0.1557518900590391,0.14754783645953137,0.1392139788332685,0.1307582701714778};
			T zGoals[] = {0.31756840547539744,0.32748579299353975,0.3378005717933035,0.3484792783644661,0.3594884491968049,0.3707946207800972,0.38236432960412026,0.39416411215865155,0.40616050493346834,0.4183200444183479,0.4306092671030677,0.44299470947740494,0.455442908031137,0.4679203992540413,0.48039371963589506,0.49282940566647565,0.5051939938355604,0.5174540206329267,0.5295760225483518,0.541526536071613,0.5532720976924878,0.5647792439007534,0.5760145111861872,0.5869444360385665,0.5975355549476685,0.607754404403259,0.617684862624283,0.6269833110602612,0.6356398015809976,0.6436439919363848,0.6509877754875409,0.6576613603932012,0.6636610682008525,0.6689826360732843,0.67361769235995,0.6775587102765505,0.6807894425694675,0.6832982853518634,0.6850685995152646,0.6860878343790213,0.6863475050397387,0.6858378423067206,0.6845542073433015,0.6824970237151333,0.679674950734998,0.6760903644476528,0.6717601780334646,0.6666986364356934,0.6609289904640975,0.6544774785385726,0.6473731907857357,0.6396420054814256,0.6313175949625012,0.6224353167682047,0.6130293234274724,0.6031408593626014,0.592806201725962,0.5820708170060365,0.5709740379170426,0.5595578699404182,0.5478698928627949,0.5359537973679934,0.5238581689279385,0.5116302430406027,0.4993208702348768,0.48697884889804044,0.4746532265736393,0.4623956012593628,0.4502485492310045,0.4382596910462214,0.42647866542842733,0.41494764320821875,0.4037154159777041,0.39282851027202587,0.3823335341683771,0.37227688282859595,0.36269856168881953,0.3536372194160615,0.3451304536909037,0.33721132075471055,0.32990808625336293,0.32324779496703726,0.3172587080115004,0.31196097533352374,0.3073723368592661,0.3035111825862362,0.3003931753833552,0.29803437903462704,0.2964420439874006,0.29562574916758594,0.2955914743255737,0.29633569503272444,0.29786307504133164,0.3001661781693752,0.30323465990169074,0.3070520940451594,0.3115998595664622,0.31685456748736623,0.3227967201975068,0.32939373258736826,0.336620401368088,0.34444137126548074,0.35282689981652365,0.3617486647489115,0.37117885795827055,0.3810878162585716,0.39144908831456365,0.4022280096944361,0.41338530657545247,0.42488278556704895,0.43667841400710394,0.4487240639652002,0.46096913859882693,0.47336419850958344,0.48584480962195276,0.4983474665337738,0.5108078008561817,0.5231640943169621,0.5353543092708679,0.5473260617454424,0.5590277695983947,0.5704169154454125,0.5814512174681081,0.5920942512854084,0.6023199392964617,0.6120999705542476,0.6214094905886763,0.6302189177557694,0.6384948605857313,0.6462095585403849,0.6533240617001391,0.6598152959296799,0.6656559786178493,0.6708180864798622,0.6752789417653723,0.6790134023420558,0.6819951301275096,0.6841972139799184,0.6856012933475145,0.6861863032005122,0.6859428344922742,0.6848659833744687,0.6829691340687087,0.6802671521438632,0.6767869940428872,0.6725559197263683,0.6675988289655357,0.6619506036376687,0.6556389703050904,0.6486908922059266,0.6411394935332501,0.6330153548211616,0.6243554553896911,0.6151955559582207,0.6055661079526033,0.5954998822170263,0.5850395750753018,0.5742180207932119,0.563083467349193,0.5516783030193103,0.5400466850891197,0.5282336225055959,0.5162846587691582,0.5042482411832561,0.49216898910506657,0.4800977462683095,0.4680759949262634,0.45614115835012375,0.4443280391401759,0.4326669709393109,0.42118572108087715,0.40991017010577024,0.3988536909103188,0.3880358943577357,0.37746325851094437,0.3677153816185399,0.3583626678734707,0.34943548878219216,0.34096421585115233,0.33297922058679913,0.3255108744955808,0.3185895490839454,0.31224561585834093,0.30650944632521565,0.30141141199101745,0.2969818843621946,0.29325123494519506,0.29024983524646697,0.28800805677245844,0.2865562710296175,0.28592484952439223,0.2861441637632308,0.28724458525258123,0.2892564854988917,0.2922102360086102,0.2961362082881848,0.30106477384406377,0.30702630418269494,0.3140511708105266,0.32216974523400665};
			int numGoals = 200; 	double tstep = totalTime/(numGoals-1);	double goalNum = time/tstep;
			double fraction = goalNum - std::floor(goalNum);				int rep = static_cast<int>(std::floor(goalNum)) / numGoals;
			int rd = static_cast<int>(std::floor(goalNum)) % numGoals;		int ru = static_cast<int>(std::ceil(goalNum)) % numGoals;
			goal[0] = (1-fraction)*xGoals[rd] + fraction*xGoals[ru];		goal[3] = 0.0;
			goal[1] = (1-fraction)*yGoals[rd] + fraction*yGoals[ru];		goal[4] = 0.0;
			goal[2] = (1-fraction)*zGoals[rd] + fraction*zGoals[ru];		goal[5] = 0.0;
			// goal[1] = goal[1] * 1.75;
			// goal[2] = goal[2] * 1.25;
			return rep;
		}

		// load initial goal
    	void loadInitialGoal(T *goal){loadFig8Goal(goal,0);}

    	// load nominal target
    	void loadInitialTarget(T *goal, T *target = nullptr){for(int i = 0; i < STATE_SIZE; i++){goal[i] = (target == nullptr) ? 0 : target[i];}}

    	// keep track of traj times
    	void newTrajCallback_f(const lcm::ReceiveBuffer *rbuf, const std::string &chan, const drake::lcmt_trajectory_f *msg){
            if (inFig8){gettimeofday(&end,NULL); timeCount++; timeTotal += time_delta_ms(start,end);} gettimeofday(&start,NULL);
        }
        void newTrajCallback_d(const lcm::ReceiveBuffer *rbuf, const std::string &chan, const drake::lcmt_trajectory_d *msg){
            if (inFig8){gettimeofday(&end,NULL); timeCount++; timeTotal += time_delta_ms(start,end);} gettimeofday(&start,NULL);
        }

		// update goal based on status
		void handleStatus(const lcm::ReceiveBuffer *rbuf, const std::string &chan, const drake::lcmt_iiwa_status *msg){
			// get current goal
			T goal[3]; double time = inFig8 ? msg->utime - zeroTime : 0; int rep = loadFig8Goal(goal,time);
			// compute the position error norm and velocity norm
			T eNorm; T vNorm; T currX[STATE_SIZE]; T eePos[NUM_POS];
			for(int i=0; i < STATE_SIZE; i++){
				if(i < NUM_POS){currX[i] = static_cast<T>(msg->joint_position_measured[i]);}
				else{			currX[i] = static_cast<T>(msg->joint_velocity_estimated[i-NUM_POS]);}
			}
			evNorm<T>(currX, goal, &eNorm, &vNorm, eePos);		totalError += static_cast<double>(eNorm);	numIters++;
			// debug print
			// printf("[%f] eNorm[%f] vNorm[%f] for goal[%f %f %f] and Pos[%f %f %f]\n",static_cast<double>(msg->utime),eNorm,vNorm,goal[0],goal[1],goal[2],eePos[0],eePos[1],eePos[2]);
			// print the error for each rep
			if(rep > currRep){
				printf("[!] Rep [%d] has total error [%f] with time [%f]\n",rep,totalError/numIters,timeTotal/timeCount); 
				totalError = 0; numIters = 0; currRep++; timeCount = 0; timeTotal = 0;
			}
			// then figure out if we are in the goal moving time
			if(inFig8){
				// then load in goal pos and zero out vel, orientation, angularVelocity (for now) -- note orientation is size 4 (quat)
				kuka::lcmt_target_twist dataOut;               dataOut.utime = msg->utime;
				for (int i = 0; i < 3; i++){dataOut.position[i] = goal[i];	dataOut.velocity[i] = 0;	
											dataOut.orientation[i] = 0;		dataOut.angular_velocity[i] = 0;}
				dataOut.orientation[3] = 0;
				// and publish it to goal channel
			    lcm_ptr.publish(ARM_GOAL_CHANNEL,&dataOut);
			}
			else {
				// else check to see if we should update goal next time
				if (eNorm < eNormLim && vNorm < vNormLim){
					// reset the zeroTime and set that we are inFig8
					zeroTime = msg->utime;		inFig8 = 1;		totalError = 0;		numIters = 0;
					// also update the solver params for this experiment
					kuka::lcmt_solver_params dataOut;	dataOut.utime = msg->utime;
					dataOut.timeLimit = timeLimit;		dataOut.iterLimit = iterLimit;		
					dataOut.clearVars = 0;              dataOut.useCostShift = 0;
					lcm_ptr.publish(SOLVER_PARAMS_CHANNEL,&dataOut);
				}
				// else if close but not there yet update the cost func to care more about moving to goals
				else if (!costSent && eNorm < 2.5*eNormLim && vNorm < 2.5*vNormLim){
					kuka::lcmt_cost_params dataOut;		dataOut.utime = msg->utime;
					dataOut.q_ee1 = _Q_EE1_fig8;		dataOut.q_ee2 = _Q_EE2_fig8;
					dataOut.qf_ee1 = _QF_EE1_fig8;		dataOut.qf_ee2 = _QF_EE2_fig8;
					dataOut.q_eev1 = _Q_EEV1_fig8;		dataOut.q_eev2 = _Q_EEV2_fig8;
					dataOut.qf_eev1 = _QF_EEV1_fig8;	dataOut.qf_eev2 = _QF_EEV2_fig8;
					dataOut.q_xdee = _Q_xdEE_fig8;		dataOut.qf_xdee = _QF_xdEE_fig8;
					dataOut.q_xee = _Q_xEE_fig8;		dataOut.qf_xee = _QF_xEE_fig8;
					dataOut.r_ee = _R_EE_fig8;			dataOut.r = _R;
					dataOut.q1 = _Q1; 					dataOut.q2 = _Q2;
					dataOut.qf1 = _QF1; 				dataOut.qf2 = _QF2;
					lcm_ptr.publish(COST_PARAMS_CHANNEL,&dataOut);
					costSent = 1;
				}
			}
			
		}
};
template <typename T>
void runFig8GoalLCM(LCM_Fig8Goal_Handler<T> *handler){
	lcm::LCM lcm_ptr; if(!lcm_ptr.good()){printf("LCM Failed to init in goal handler\n");}
	lcm::Subscription *sub = lcm_ptr.subscribe(ARM_STATUS_FILTERED, &LCM_Fig8Goal_Handler<T>::handleStatus, handler); lcm::Subscription *sub2;
	if (std::is_same<T, float>::value){sub2 = lcm_ptr.subscribe(ARM_TRAJ_CHANNEL, &LCM_Fig8Goal_Handler<T>::newTrajCallback_f, handler);}
    else if (std::is_same<T, double>::value){sub2 = lcm_ptr.subscribe(ARM_TRAJ_CHANNEL, &LCM_Fig8Goal_Handler<T>::newTrajCallback_d, handler);}
    else{printf("Timing only defined for floats and doubles\n");}
    sub->setQueueCapacity(1); sub2->setQueueCapacity(1);
    while(0 == lcm_ptr.handle());
    // while(1){lcm_ptr.handle();usleep(5000);}
}

template <typename T>
__host__
int runMPC_LCM(char mode, T *xInit){
	// launch the simulator
    // printf("Make sure the drake kuka simulator or kuka hardware is launched!!!\n");
	// get the max iters and time per solve
	printf("[For the initial step] What is the maximum number of iterations a solver can take? (q to exit)?\n");
	int itersToDo_init = getInt(1000, 1);
	// printf("[For the initial step] What should the MPC time budget be (in ms)? (q to exit)?\n");
	int timeLimit_init = 1000; //getInt(1000, 1); //note in ms
	printf("[For the figure eight] What is the maximum number of iterations a solver can take? (q to exit)?\n");
	int itersToDo = getInt(1000, 1);
	// printf("[For the figure eight] What should the MPC time budget be (in ms)? (q to exit)?\n");
	int timeLimit = 10000; //getInt(1000, 1); //note in ms
	// get the total traj time
	printf("How many seconds long should one figure eight of the tracked trajectory be? (q to exit)\n");
	double totalTime_us = 1000000.0*static_cast<double>(getInt(100, 1));
	// allocate variables and load inital variables
	trajVars<T> *tvars = new trajVars<T>; matDimms *dimms = new matDimms; algTrace<T> *atrace = new algTrace<T>;
	costParams<T> *cst = new costParams<T>;	loadCost(cst); // load in default cost to start
    std::thread mpcThread; LCM_MPCLoop_Handler<T> *mpchandler; CPUVars<T> *cvars; GPUVars<T> *gvars; // pointers for reference later
    // allocate for CPU / GPU
    if (mode == 'G'){gvars = new GPUVars<T>; allocateMemory_GPU_MPC<T>(gvars, dimms, tvars);}
    else{		     cvars = new CPUVars<T>; allocateMemory_CPU_MPC<T>(cvars, dimms, tvars);}
    // get the goal handler
    LCM_Fig8Goal_Handler<T> *goalhandler = new LCM_Fig8Goal_Handler<T>(totalTime_us, E_NORM_LIM, V_NORM_LIM, itersToDo, timeLimit);
    // then load the goals and LCM handlers and launch the MPC threads
    if (mode == 'G'){
    	// load initial traj and goal and run to full convergence to warm start
    	loadTraj<T>(gvars, tvars, dimms, xInit);	goalhandler->loadInitialGoal(gvars->xGoal);		goalhandler->loadInitialTarget(gvars->xTarget,xInit);
    	runiLQR_MPC_GPU<T>(tvars,gvars,dimms,atrace,cst,0,0,1);
		// then create the handler and launch the MPC thread
		mpchandler = new LCM_MPCLoop_Handler<T>(gvars,tvars,dimms,atrace,cst,itersToDo_init,timeLimit_init);
     	mpcThread  = std::thread(&runMPCHandler<T>, mpchandler);    
    }
    else{
    	// load initial goal and run to full convergence to warm start
    	loadTraj<T>(cvars, tvars, dimms, xInit);	goalhandler->loadInitialGoal(cvars->xGoal);		goalhandler->loadInitialTarget(cvars->xTarget,xInit);
    	runiLQR_MPC_CPU<T>(tvars,cvars,dimms,atrace,cst,0,0,1);
		// then create the handler and launch the MPC thread
		mpchandler = new LCM_MPCLoop_Handler<T>(cvars,tvars,dimms,atrace,cst,itersToDo_init,timeLimit_init);
     	mpcThread  = std::thread(&runMPCHandler<T>, mpchandler);   
     	if(FORCE_CORE_SWITCHES){setCPUForThread(&mpcThread, 1);} // move to another CPU
    }
    // launch the goal monitor
    std::thread goalThread = std::thread(&runFig8GoalLCM<T>, goalhandler);
    // launch the trajRunner
    std::thread trajThread = std::thread(&runTrajRunner<T>, dimms, TRAJ_RUNNER_ALPHA);
    // launch the status filter if needed
    #if USE_VELOCITY_FILTER
    	std::thread filterThread = std::thread(&run_IIWA_STATUS_filter<T>);
	#endif
    printf("All threads launched -- check simulator/hardware output!\n");
    mpcThread.join();	trajThread.join();	goalThread.join();
    #if USE_VELOCITY_FILTER
    	filterThread.join();
    #endif
    if (mode == 'G'){freeMemory_GPU_MPC<T>(gvars); delete gvars;} else{freeMemory_CPU_MPC<T>(cvars); delete cvars;}
    freeTrajVars<T>(tvars); delete tvars; delete atrace; delete dimms; delete cst; delete mpchandler; delete goalhandler;
    return 0;
}

int main(int argc, char *argv[])
{
	// init rand
	srand(time(NULL));
	// initial state for example
	algType xInit[STATE_SIZE]; loadInitialState(xInit,1);
	// require user input for mode of operation
	char mode = '?'; if (argc > 1){mode = argv[1][0];}
	// run the MPC loop
	if (mode == 'C' || mode == 'G'){return runMPC_LCM<algType>(mode,xInit);}
	// run aditional example options (printers, simulator, etc.)
	else{return runOtherOptions<algType>(mode,xInit,argv);}
}